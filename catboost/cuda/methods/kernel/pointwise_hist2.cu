#include "hip/hip_runtime.h"
#include "pointwise_hist2.cuh"
#include "split_properties_helpers.cuh"

#include <catboost/cuda/cuda_lib/kernel/arch.cuh>
#include <catboost/cuda/cuda_util/kernel/instructions.cuh>
#include <catboost/cuda/cuda_util/kernel/kernel_helpers.cuh>
#include <catboost/cuda/cuda_lib/kernel/arch.cuh>
#include <cstdlib>


namespace NKernel {

    template <int OUTER_HIST_BITS_COUNT, int INNER_HIST_BITS_COUNT, int BLOCK_SIZE>
    struct TPointHist {
        volatile float* Buffer;
        int BlockId;

        __forceinline__ __device__ int SliceOffset() {
            const int warpOffset = 1024 * (threadIdx.x / 32);
            const int blocks = 4  >> INNER_HIST_BITS_COUNT;
            const int innerHistStart = (threadIdx.x & ((blocks - 1) << (INNER_HIST_BITS_COUNT + 3)));
            return warpOffset + innerHistStart;
        }

        __device__ TPointHist(float* buff)
        {
            const int HIST_SIZE = 32 * BLOCK_SIZE;
            for (int i = threadIdx.x; i < HIST_SIZE; i += BLOCK_SIZE)
                buff[i] = 0;
            __syncthreads();

            Buffer = buff + SliceOffset();
            BlockId = (threadIdx.x / 32) & ((1 << OUTER_HIST_BITS_COUNT) - 1);
        }

        __device__ void AddPoint(ui32 ci, const float t, const float w) {
            const bool flag = threadIdx.x & 1;

#pragma unroll
            for (int i = 0; i < 4; i++) {
                short f = (threadIdx.x + (i << 1)) & 6;
                short bin = bfe(ci, 24 - (f << 2), 8);
                short pass = (bin >> (5 + INNER_HIST_BITS_COUNT)) == BlockId;
                int offset0 = f + flag;
                int offset1 = f + !flag;

                const int mask = (1 << INNER_HIST_BITS_COUNT) - 1;

                const int tmp = (((bin >> INNER_HIST_BITS_COUNT) & 31) << 5) + 8 * (bin & mask);
                offset0 += tmp;
                offset1 += tmp;

                if (INNER_HIST_BITS_COUNT > 0)
                {
#pragma unroll
                    for (int k = 0; k < (1 << INNER_HIST_BITS_COUNT); ++k)
                    {
                        if (((threadIdx.x >> 3) & ((1 << INNER_HIST_BITS_COUNT) - 1)) == k)
                        {
                            Buffer[offset0] += (flag ? t : w) * pass;
                            Buffer[offset1] += (flag ? w : t) * pass;
                        }
                    }
                } else {
                    Buffer[offset0] += (flag ? t : w) * pass;
                    Buffer[offset1] += (flag ? w : t) * pass;
                }
            }
        }

        //After reduce we store histograms by blocks: 256 floats (4 x 2 x 32)
        // for first 32 bins; than 256 floats for second 32 bins, etc
        __device__ void Reduce()
        {

            Buffer -= SliceOffset();

            const int innerHistCount = 4 >> INNER_HIST_BITS_COUNT;
            const int warpCount = BLOCK_SIZE >> 5;
            const int warpHistCount = warpCount >> OUTER_HIST_BITS_COUNT;
            const int fold = (threadIdx.x >> 3) & 31;

            const int mask = (1 << INNER_HIST_BITS_COUNT) - 1;
            const int binOffset = ((fold >> INNER_HIST_BITS_COUNT) << 5) + 8 * (fold & mask);
            const int offset = (threadIdx.x & 7) + binOffset;


#pragma unroll
            for (int outerBits = 0; outerBits < 1 << (OUTER_HIST_BITS_COUNT); ++outerBits)
            {
                for (int innerBits = 0; innerBits < (1 << (INNER_HIST_BITS_COUNT)); ++innerBits)
                {
                    float sum = 0.0;

                    const int innerOffset = innerBits << (10 - INNER_HIST_BITS_COUNT);
                    if (threadIdx.x < 256)
                    {
#pragma unroll
                        for (int hist = 0; hist < warpHistCount; ++hist)
                        {
                            const int warpOffset = ((hist << OUTER_HIST_BITS_COUNT) + outerBits) * 1024;

#pragma unroll
                            for (int inWarpHist = 0; inWarpHist < innerHistCount; ++inWarpHist)
                            {
                                sum += Buffer[offset + warpOffset + innerOffset +
                                              (inWarpHist << (3 + INNER_HIST_BITS_COUNT))];
                            }
                        }
                    }
                    __syncthreads();

                    if (threadIdx.x < 256)
                    {
                        Buffer[threadIdx.x + 256 * (innerBits | (outerBits << INNER_HIST_BITS_COUNT))] = sum;
                    }
                }
            }
            __syncthreads();
        }
    };

    template <int STRIPE_SIZE, int HIST_BLOCK_COUNT, int N, int BLOCKS_PER_FEATURE, typename THist>
    __forceinline__ __device__ void ComputeHistogram(
            const ui32* __restrict indices, int dsSize,
            const float* __restrict target, const float* __restrict weight,
            const ui32* __restrict cindex, float* result)
    {

        indices += (blockIdx.x % BLOCKS_PER_FEATURE) * STRIPE_SIZE;
        target += (blockIdx.x % BLOCKS_PER_FEATURE) * STRIPE_SIZE;
        weight += (blockIdx.x % BLOCKS_PER_FEATURE) * STRIPE_SIZE;
        dsSize = max(dsSize - (blockIdx.x % BLOCKS_PER_FEATURE) * STRIPE_SIZE, 0);
        const int stripe = STRIPE_SIZE * BLOCKS_PER_FEATURE;

        THist hist(result);
        if (dsSize)
        {
            int i = (threadIdx.x & 31) + (threadIdx.x / 32 / HIST_BLOCK_COUNT) * 32;
            int iteration_count = (dsSize - i + (stripe - 1)) / stripe;
            int blocked_iteration_count = ((dsSize - (i | 31) + (stripe - 1)) / stripe) / N;

            weight += i;
            target += i;
            indices += i;

#pragma unroll 4
            for (int j = 0; j < blocked_iteration_count; ++j)
            {
                ui32 local_index[N];
#pragma unroll
                for (int k = 0; k < N; k++)
                {
                    local_index[k] = __ldg(indices + stripe * k);
                }

                ui32 local_ci[N];
                float local_w[N];
                float local_wt[N];

#pragma unroll
                for (int k = 0; k < N; ++k)
                {
                    local_ci[k] = __ldg(cindex + local_index[k]);
                    local_w[k] = __ldg(weight + stripe * k);
                    local_wt[k] = __ldg(target + stripe * k);
                }

#pragma unroll
                for (int k = 0; k < N; ++k)
                {
                    hist.AddPoint(local_ci[k], local_wt[k], local_w[k]);
                }

                i += stripe * N;
                indices += stripe * N;
                target += stripe * N;
                weight += stripe * N;
            }

            for (int k = blocked_iteration_count * N; k < iteration_count; ++k)
            {
                const int index = __ldg(indices);
                ui32 ci = __ldg(cindex + index);
                float w = __ldg(weight);
                float wt = __ldg(target);
                hist.AddPoint(ci, wt, w);
                i += stripe;
                indices += stripe;
                target += stripe;
                weight += stripe;
            }
            __syncthreads();

            hist.Reduce();
        }
    }



    template <int BLOCK_SIZE, int OUTER_HIST_BITS_COUNT, int INNER_HIST_BITS_COUNT, int N, int BLOCKS_PER_FEATURE>
    __forceinline__ __device__ void ComputeSplitPropertiesPass(const TCFeature* __restrict feature, const ui32* __restrict cindex,
                                                               const float* __restrict target, const float* __restrict weight, const ui32* __restrict indices,
                                                               const TDataPartition* __restrict partition, int fCount,
                                                               float* binSumsForPart,
                                                               float* smem) {

        using THist = TPointHist < OUTER_HIST_BITS_COUNT, INNER_HIST_BITS_COUNT, BLOCK_SIZE >;
        const int stripeSize = BLOCK_SIZE >> OUTER_HIST_BITS_COUNT;
        const int histBlockCount =  1 << OUTER_HIST_BITS_COUNT;

        ComputeHistogram<stripeSize, histBlockCount, N,  BLOCKS_PER_FEATURE, THist >(indices + partition->Offset,
                partition->Size, target + partition->Offset, weight + partition->Offset, cindex, smem);

        __syncthreads();



        int fid = (threadIdx.x / 64);
        int fold = (threadIdx.x / 2) & 31;


        for (int upperBits = 0; upperBits < (1 << (OUTER_HIST_BITS_COUNT + INNER_HIST_BITS_COUNT)); ++upperBits) {
            const int binOffset = upperBits << 5;

            if (fid < fCount && fold < min((int)feature[fid].Folds - binOffset, 32)) {
                int w = threadIdx.x & 1;
                if (BLOCKS_PER_FEATURE > 1) {
                    atomicAdd(binSumsForPart + (feature[fid].FirstFoldIndex + fold + binOffset) * 2 + w, smem[fold * 8 + 2 * fid + w + 256 * upperBits]);
                } else {
                    binSumsForPart[(feature[fid].FirstFoldIndex + fold + binOffset) * 2 + w] = smem[fold * 8 + 2 * fid + w + 256 * upperBits];
                }
            }
        }

        __syncthreads();


    }



#define DECLARE_PASS(O, I, N, M) \
    ComputeSplitPropertiesPass<BLOCK_SIZE, O, I, N, M>(feature, cindex, target, weight, indices, partition, fCount, binSums, &counters[0]);


    template <int BLOCK_SIZE, bool FULL_PASS, int M>
#if __CUDA_ARCH__ >= 520
    __launch_bounds__(BLOCK_SIZE, 2)
#else
    __launch_bounds__(BLOCK_SIZE, 1)
#endif
    __global__ void ComputeSplitPropertiesNBImpl(
            const TCFeature* __restrict feature, int fCount, const ui32* __restrict cindex,
            const float* __restrict target, const float* __restrict weight, int dsSize,
            const ui32* __restrict indices,
            const TDataPartition* partition,
            float* binSums,
            const int totalFeatureCount) {


        TPartOffsetsHelper helper(gridDim.z);
        helper.ShiftPartAndBinSumsPtr(partition, binSums, totalFeatureCount, FULL_PASS);

        feature += (blockIdx.x / M) * 4;
        cindex += feature->Offset * ((size_t)dsSize);
        fCount = min(fCount - (blockIdx.x / M)  * 4, 4);

//
        __shared__ float counters[32 * BLOCK_SIZE];
        const int maxBinCount = GetMaxBinCount(feature, fCount, (int*) &counters[0]);
        __syncthreads();


        if (partition->Size) {
            if (maxBinCount <= 32) {
                DECLARE_PASS(0, 0, 8,  M);
            }
            else if (maxBinCount <= 64) {
                DECLARE_PASS(0, 1, 4, M);
            } else if (maxBinCount <= 128) {
                DECLARE_PASS(0, 2, 4, M);
            } else {
                DECLARE_PASS(1, 2, 4, M);
            }
        }
    }



    template <int BLOCK_SIZE>
    struct TPointHistHalfByte {
        volatile float* Buffer;

        __forceinline__ __device__ int SliceOffset() {
            const int warpOffset = 512 * (threadIdx.x / 32);
            const int innerHistStart =  threadIdx.x & 16;
            return warpOffset + innerHistStart;
        }

        __device__  TPointHistHalfByte(float* buff)
        {
            const int HIST_SIZE = 16 * BLOCK_SIZE;
            for (int i = threadIdx.x; i < HIST_SIZE; i += BLOCK_SIZE)
                buff[i] = 0;
            __syncthreads();

            Buffer = buff + SliceOffset();
        }

        __device__ void AddPoint(ui32 ci, const float t, const float w) {

            const bool flag = threadIdx.x & 1;

#pragma unroll
            for (int i = 0; i < 8; i++) {
                const short f = (threadIdx.x + (i << 1)) & 14;
                short bin = bfe(ci, 28 - (f << 1), 4);
                bin <<= 5;
                bin += f;
                const int offset0 = bin + flag;
                const int offset1 = bin + !flag;
                Buffer[offset0] += flag ? t : w;
                Buffer[offset1] += flag ? w : t;
            }
        }

        __device__ void Reduce()
        {
            Buffer -= SliceOffset();

            {
                const int warpCount = BLOCK_SIZE >> 5;
                const int fold = (threadIdx.x >> 5) & 15;
                const int sumOffset = threadIdx.x & 31;


                float sum = 0.0;
                if (threadIdx.x < 512)
                {
                    #pragma unroll
                    for (int warpId = 0; warpId < warpCount; ++warpId)
                    {
                        const int warpOffset = 512 * warpId;
                        sum += Buffer[warpOffset + sumOffset + 32 * fold];
                    }
                }
                __syncthreads();

                if (threadIdx.x < 512)
                {
                    Buffer[threadIdx.x] = sum;
                }
            }

            __syncthreads();
            const int fold = (threadIdx.x >> 4) & 15;
            float sum = 0.0f;

            if (threadIdx.x < 256) {
                const int histEntryId = (threadIdx.x & 15);
                sum = Buffer[32 * fold + histEntryId] + Buffer[32 * fold + histEntryId + 16];
            }

            __syncthreads();

            if (threadIdx.x < 256) {
                Buffer[threadIdx.x] = sum;
            }

            __syncthreads();
        }
    };

    inline constexpr __device__ __host__ int GetUnrollFactorHalfByteHist() {
        #if __CUDA_ARCH__ >= 600
        return 1;
        #elif __CUDA_ARCH__ >= 520
        return 2;
        #else
        return 4;
        #endif
    }

    template <int BLOCK_SIZE,  bool FULL_PASS, int M>
    #if __CUDA_ARCH__ >= 520
    __launch_bounds__(BLOCK_SIZE, 2)
    #else
    __launch_bounds__(BLOCK_SIZE, 1)
    #endif
    __global__ void ComputeSplitPropertiesBImpl(
            const TCFeature* __restrict feature, int fCount, const ui32* __restrict cindex,
            const float* __restrict target, const float* __restrict weight, int dsSize, const ui32* __restrict indices,
            const TDataPartition* partition, float* binSums, int totalFeatureCount)
    {

        TPartOffsetsHelper helper(gridDim.z);
        helper.ShiftPartAndBinSumsPtr(partition, binSums, totalFeatureCount, FULL_PASS);

        feature += (blockIdx.x / M) * 32;
        cindex += feature->Offset * ((size_t)dsSize);
        fCount = min(fCount - (blockIdx.x / M) * 32, 32);

        __shared__ float counters[16 * BLOCK_SIZE];

        if (partition->Size)
        {

            using THist = TPointHistHalfByte<BLOCK_SIZE>;

            ComputeHistogram < BLOCK_SIZE, 1, GetUnrollFactorHalfByteHist(), M, THist > (indices + partition->Offset,
                    partition->Size, target + partition->Offset, weight + partition->Offset, cindex, &counters[0]);

            ui32 w = threadIdx.x & 1;
            ui32 fid = (threadIdx.x >> 1);

            if (fid < fCount)
            {
                const int groupId = fid / 4;
                uchar fMask = 1 << (3 - (fid & 3));

                float sum = 0.f;
                #pragma uroll
                for (int i = 0; i < 16; i++) {
                    if (!(i & fMask))
                        sum += counters[i * 16 + 2 * groupId + w];
                }

                if (M > 1) {
                    atomicAdd(binSums + (feature[fid].FirstFoldIndex) * 2 + w, sum);
                } else {
                    binSums[(feature[fid].FirstFoldIndex) * 2 + w] = sum;
                }
            }
        }
    }

    template <int BLOCK_SIZE,
              int BLOCKS_PER_FEATURE_COUNT>
    inline void RunComputeHist2NonBinaryKernel(const TCFeature* nbFeatures, int nbCount,
                                              const ui32* cindex, int dsSize,
                                              const float* target, const float* weight,  const ui32* indices,
                                              const TDataPartition* partition,
                                              float* binSums, const int binFeatureCount,
                                              bool fullPass,
                                              TCudaStream stream,
                                              dim3 numBlocks) {

        if (fullPass)
        {
            ComputeSplitPropertiesNBImpl < BLOCK_SIZE, true, BLOCKS_PER_FEATURE_COUNT > << <numBlocks, BLOCK_SIZE, 0, stream>>>(
                    nbFeatures, nbCount, cindex, target, weight, dsSize,
                            indices, partition, binSums, binFeatureCount
            );

        } else {
                ComputeSplitPropertiesNBImpl < BLOCK_SIZE, false, BLOCKS_PER_FEATURE_COUNT> << <numBlocks, BLOCK_SIZE, 0, stream>>>(
                        nbFeatures, nbCount, cindex, target, weight, dsSize,
                                indices, partition, binSums, binFeatureCount);
        }

    }

    inline ui32 EstimateBlockPerFeatureMultiplier(dim3 numBlocks, ui32 dsSize) {
        ui32 multiplier = 1;
        while ((numBlocks.x * numBlocks.y * min(numBlocks.z, 4) * multiplier < TArchProps::SMCount()) && ((dsSize / multiplier) > 10000) && (multiplier < 64)) {
            multiplier *= 2;
        }
        return multiplier;
    }

    void ComputeHist2NonBinary(const TCFeature* nbFeatures, int nbCount,
                               const ui32* cindex, int dsSize,
                               const float* target, const float* weight,  const ui32* indices,
                               const TDataPartition* partition, ui32 partCount, ui32 foldCount,
                               float* binSums, const int binFeatureCount,
                               bool fullPass,
                               TCudaStream stream)
    {
        if (nbCount) {
            dim3 numBlocks;
            numBlocks.x = (nbCount + 3) / 4;
            const int histPartCount = (fullPass ? partCount : partCount / 2);
            numBlocks.y = histPartCount;
            numBlocks.z = foldCount;

            const int blockSize = 384;
            const ui32 multiplier = min(EstimateBlockPerFeatureMultiplier(numBlocks, dsSize), 64);
            numBlocks.x *= multiplier;

            if (multiplier == 1) {
                RunComputeHist2NonBinaryKernel<blockSize, 1>(nbFeatures, nbCount, cindex, dsSize, target, weight, indices, partition, binSums, binFeatureCount, fullPass, stream, numBlocks);
            } else if (multiplier == 2) {
                RunComputeHist2NonBinaryKernel<blockSize, 2>(nbFeatures, nbCount, cindex, dsSize, target, weight, indices, partition, binSums, binFeatureCount, fullPass, stream, numBlocks);
            } else if (multiplier == 4) {
                RunComputeHist2NonBinaryKernel<blockSize, 4>(nbFeatures, nbCount, cindex, dsSize, target, weight, indices, partition, binSums, binFeatureCount, fullPass, stream, numBlocks);
            } else if (multiplier == 8) {
                RunComputeHist2NonBinaryKernel<blockSize, 8>(nbFeatures, nbCount, cindex, dsSize, target, weight, indices, partition, binSums, binFeatureCount, fullPass, stream, numBlocks);
            } else if (multiplier == 16) {
                RunComputeHist2NonBinaryKernel<blockSize, 16>(nbFeatures, nbCount, cindex, dsSize, target, weight, indices, partition, binSums, binFeatureCount, fullPass, stream, numBlocks);
            } else if (multiplier == 32) {
                RunComputeHist2NonBinaryKernel<blockSize, 32>(nbFeatures, nbCount, cindex, dsSize, target, weight, indices, partition, binSums, binFeatureCount, fullPass, stream, numBlocks);
            } else if (multiplier == 64) {
                RunComputeHist2NonBinaryKernel<blockSize, 64>(nbFeatures, nbCount, cindex, dsSize, target, weight, indices, partition, binSums, binFeatureCount, fullPass, stream, numBlocks);
            } else {
                exit(1);
            }

            const int scanBlockSize = 256;
            dim3 scanBlocks;
            scanBlocks.x = (nbCount * 32 + scanBlockSize - 1) / scanBlockSize;
            scanBlocks.y = histPartCount;
            scanBlocks.z = foldCount;
            const int scanOffset = fullPass ? 0 : ((partCount / 2) * binFeatureCount * 2) * foldCount;
            ScanHistogramsImpl<scanBlockSize, 2><<<scanBlocks, scanBlockSize, 0, stream>>>(nbFeatures, nbCount, binFeatureCount, binSums + scanOffset);

            if (!fullPass) {
                UpdatePointwiseHistograms(binSums,  binFeatureCount, partCount, foldCount, 2, partition, stream);
            }
        }
    }

    template <int BLOCK_SIZE, int BLOCKS_PER_FEATURE_COUNT>
    void RunComputeHist2BinaryKernel(const TCFeature* bFeatures, int bCount,
                                     const ui32* cindex, int dsSize,
                                     const float* target, const float* weight, const ui32* indices,
                                     const TDataPartition* partition,
                                     float* binSums, bool fullPass,
                                     TCudaStream stream,
                                     dim3 numBlocks) {
        if (fullPass)
        {
            ComputeSplitPropertiesBImpl < BLOCK_SIZE, true, BLOCKS_PER_FEATURE_COUNT > << <numBlocks, BLOCK_SIZE, 0, stream>>>(
                    bFeatures, bCount, cindex, target, weight, dsSize,
                            indices, partition, binSums, bCount
            );
        } else {
            ComputeSplitPropertiesBImpl < BLOCK_SIZE, false, BLOCKS_PER_FEATURE_COUNT > << <numBlocks, BLOCK_SIZE, 0, stream>>>(
                    bFeatures, bCount, cindex, target, weight, dsSize,
                            indices, partition, binSums, bCount
            );
        }
    };

    void ComputeHist2Binary(const TCFeature* bFeatures, int bCount,
                            const ui32* cindex, int dsSize,
                            const float* target, const float* weight, const ui32* indices,
                            const TDataPartition* partition, ui32 partsCount, ui32 foldCount,
                            float* binSums, bool fullPass,
                            TCudaStream stream)
    {
        dim3 numBlocks;
        numBlocks.x = (bCount + 31) / 32;
        const int histCount = fullPass ? partsCount : partsCount / 2;
        numBlocks.y = histCount;
        numBlocks.z = foldCount;

        const int blockSize = 768;
        const ui32 multiplier = min(EstimateBlockPerFeatureMultiplier(numBlocks, dsSize), 64);
        numBlocks.x *= multiplier;

        if (bCount) {

            if (multiplier == 1) {
                RunComputeHist2BinaryKernel<blockSize, 1>(bFeatures, bCount, cindex, dsSize, target, weight, indices, partition, binSums, fullPass, stream, numBlocks);
            } else if (multiplier == 2) {
                RunComputeHist2BinaryKernel<blockSize, 2>(bFeatures, bCount, cindex, dsSize, target, weight, indices, partition, binSums, fullPass, stream, numBlocks);
            } else if (multiplier == 4) {
                RunComputeHist2BinaryKernel<blockSize, 4>(bFeatures, bCount, cindex, dsSize, target, weight, indices, partition,  binSums, fullPass, stream, numBlocks);
            } else if (multiplier == 8) {
                RunComputeHist2BinaryKernel<blockSize, 8>(bFeatures, bCount, cindex, dsSize, target, weight, indices, partition, binSums, fullPass, stream, numBlocks);
            } else if (multiplier == 16) {
                RunComputeHist2BinaryKernel<blockSize, 16>(bFeatures, bCount, cindex, dsSize, target, weight, indices, partition, binSums, fullPass, stream, numBlocks);
            } else if (multiplier == 32) {
                RunComputeHist2BinaryKernel<blockSize, 32>(bFeatures, bCount, cindex, dsSize, target, weight, indices, partition, binSums, fullPass, stream, numBlocks);
            } else if (multiplier == 64) {
                RunComputeHist2BinaryKernel<blockSize, 64>(bFeatures, bCount, cindex, dsSize, target, weight, indices, partition, binSums, fullPass, stream, numBlocks);
            } else {
                exit(1);
            }

            if (!fullPass) {
                UpdatePointwiseHistograms(binSums, bCount, partsCount, foldCount, 2, partition, stream);
            }
        }
    }



    template <int BLOCK_SIZE, bool FULL_PASS, int M>
#if __CUDA_ARCH__ >= 520
    __launch_bounds__(BLOCK_SIZE, 2)
#else
    __launch_bounds__(BLOCK_SIZE, 1)
#endif
    __global__ void ComputeSplitPropertiesHalfByteImpl(
            const TCFeature* __restrict feature, int fCount, const ui32* __restrict cindex,
            const float* __restrict target, const float* __restrict weight, int dsSize,
            const ui32* __restrict indices,
            const TDataPartition* partition,
            float* binSums,
            const int totalFeatureCount) {


        TPartOffsetsHelper helper(gridDim.z);
        helper.ShiftPartAndBinSumsPtr(partition, binSums, totalFeatureCount, FULL_PASS);

        feature += (blockIdx.x / M) * 8;
        cindex += feature->Offset * ((size_t)dsSize);
        fCount = min(fCount - (blockIdx.x / M) * 8, 8);

//
        __shared__ float smem[16 * BLOCK_SIZE];


        using THist = TPointHistHalfByte<BLOCK_SIZE>;


        ComputeHistogram<BLOCK_SIZE, 1, GetUnrollFactorHalfByteHist(),  M, THist >(indices + partition->Offset, partition->Size,
                                                       target + partition->Offset, weight + partition->Offset,
                                                       cindex, smem);

        __syncthreads();

        const int fid = (threadIdx.x / 32);
        const int fold = (threadIdx.x / 2) & 15;
        const int w = threadIdx.x & 1;


        if (fid < fCount && fold < feature[fid].Folds) {
            if (M > 1) {
                atomicAdd(binSums + (feature[fid].FirstFoldIndex + fold) * 2 +  w, smem[fold * 16 + 2 * fid + w]);
            } else {
                binSums[(feature[fid].FirstFoldIndex + fold) * 2 +  w] = smem[fold * 16 + 2 * fid + w];
            }
        }

    }


    template <int BLOCK_SIZE,
              int BLOCKS_PER_FEATURE_COUNT>
    inline void RunComputeHist2HalfByteKernel(const TCFeature* nbFeatures, int nbCount,
                                              const ui32* cindex, int dsSize,
                                              const float* target, const float* weight,  const ui32* indices,
                                              const TDataPartition* partition,
                                              float* binSums, const int binFeatureCount,
                                              bool fullPass,
                                              TCudaStream stream,
                                              dim3 numBlocks) {

        if (fullPass)
        {
            ComputeSplitPropertiesHalfByteImpl < BLOCK_SIZE, true, BLOCKS_PER_FEATURE_COUNT > << <numBlocks, BLOCK_SIZE, 0, stream>>>(
                    nbFeatures, nbCount, cindex, target, weight, dsSize,
                            indices, partition, binSums, binFeatureCount
            );

        } else {
            ComputeSplitPropertiesHalfByteImpl < BLOCK_SIZE, false, BLOCKS_PER_FEATURE_COUNT> << <numBlocks, BLOCK_SIZE, 0, stream>>>(
                    nbFeatures, nbCount, cindex, target, weight, dsSize,
                            indices, partition, binSums, binFeatureCount);
        }

    }

    void ComputeHist2HalfByte(const TCFeature* halfByteFeatures, int halfByteFeaturesCount,
                              const ui32* cindex, int dsSize,
                              const float* target, const float* weight,  const ui32* indices,
                              const TDataPartition* partition, ui32 partsCount, ui32 foldCount,
                              float* binSums, const int binFeatureCount,
                              bool fullPass,
                              TCudaStream stream)
    {
        dim3 numBlocks;
        numBlocks.x = static_cast<ui32>((halfByteFeaturesCount + 7) / 8);
        const int histCount = fullPass ? partsCount : partsCount / 2;
        numBlocks.y = static_cast<ui32>(histCount);
        numBlocks.z = foldCount;

        const int blockSize = 768;
        const ui32 multiplier = min(EstimateBlockPerFeatureMultiplier(numBlocks, dsSize), 64);
        numBlocks.x *= multiplier;

        if (halfByteFeaturesCount) {

            if (multiplier == 1) {
                RunComputeHist2HalfByteKernel<blockSize, 1>(halfByteFeatures, halfByteFeaturesCount, cindex, dsSize, target, weight, indices, partition, binSums, binFeatureCount, fullPass, stream, numBlocks);
            } else if (multiplier == 2) {
                RunComputeHist2HalfByteKernel<blockSize, 2>(halfByteFeatures, halfByteFeaturesCount, cindex, dsSize, target, weight, indices, partition, binSums, binFeatureCount, fullPass, stream, numBlocks);
            } else if (multiplier == 4) {
                RunComputeHist2HalfByteKernel<blockSize, 4>(halfByteFeatures, halfByteFeaturesCount, cindex, dsSize, target, weight, indices, partition,  binSums, binFeatureCount,  fullPass, stream, numBlocks);
            } else if (multiplier == 8) {
                RunComputeHist2HalfByteKernel<blockSize, 8>(halfByteFeatures, halfByteFeaturesCount, cindex, dsSize, target, weight, indices, partition, binSums, binFeatureCount, fullPass, stream, numBlocks);
            } else if (multiplier == 16) {
                RunComputeHist2HalfByteKernel<blockSize, 16>(halfByteFeatures, halfByteFeaturesCount, cindex, dsSize, target, weight, indices, partition, binSums, binFeatureCount,  fullPass, stream, numBlocks);
            } else if (multiplier == 32) {
                RunComputeHist2HalfByteKernel<blockSize, 32>(halfByteFeatures, halfByteFeaturesCount, cindex, dsSize, target, weight, indices, partition, binSums, binFeatureCount,  fullPass, stream, numBlocks);
            } else if (multiplier == 64) {
                RunComputeHist2HalfByteKernel<blockSize, 64>(halfByteFeatures, halfByteFeaturesCount, cindex, dsSize, target, weight, indices, partition, binSums, binFeatureCount,  fullPass, stream, numBlocks);
            } else {
                exit(1);
            }

            const int scanBlockSize = 256;
            dim3 scanBlocks;
            scanBlocks.x = static_cast<ui32>((halfByteFeaturesCount * 32 + scanBlockSize - 1) / scanBlockSize);
            scanBlocks.y = static_cast<ui32>(histCount);
            scanBlocks.z = foldCount;
            const int scanOffset = fullPass ? 0 : ((partsCount / 2) * binFeatureCount * 2) * foldCount;
            ScanHistogramsImpl<scanBlockSize, 2><<<scanBlocks, scanBlockSize, 0, stream>>>(halfByteFeatures, halfByteFeaturesCount, binFeatureCount, binSums + scanOffset);

            if (!fullPass) {
                UpdatePointwiseHistograms(binSums,  binFeatureCount, partsCount, foldCount, 2, partition, stream);
            }
        }
    }




    __global__ void UpdateBinsImpl(ui32* dstBins, const ui32* bins, const ui32* docIndices, ui32 size,
                                   ui32 loadBit, ui32 foldBits) {
        const ui32 i = blockIdx.x * blockDim.x + threadIdx.x;

        if (i < size) {
            const ui32 idx = LdgWithFallback(docIndices, i);
            const ui32 bit = (LdgWithFallback(bins, idx) >> loadBit) & 1;
            dstBins[i] =  dstBins[i] | (bit << (loadBit + foldBits));
        }
    }

    void UpdateFoldBins(ui32* dstBins, const ui32* bins, const ui32* docIndices, ui32 size,
                        ui32 loadBit, ui32 foldBits, TCudaStream stream) {


        const ui32 blockSize = 256;
        const ui32 numBlocks = CeilDivide(size, blockSize);
        UpdateBinsImpl<<<numBlocks, blockSize, 0, stream>>>(dstBins, bins, docIndices, size, loadBit, foldBits);
    }

}
