#include "hip/hip_runtime.h"
#include "random.cuh"
#include "random_gen.cuh"
#include "catboost/cuda/cuda_lib/kernel/arch.cuh"

namespace NKernel {

    __global__ void PoissonRandImpl(ui64* seeds, ui32 seedSize,
                                    const float* alpha, int* result)
    {
        ui32 i = blockIdx.x * blockDim.x + threadIdx.x;
        while (i < seedSize) {
            ui64 s = seeds[i];
            result[i] = NextPoisson(&s, alpha[i]);
            seeds[i] = s;
            i += gridDim.x * blockDim.x;
        }
    }

    void PoissonRand(ui64* seeds, ui32 size, const float* alphas, int* result, TCudaStream stream)
    {
        const ui32 blockSize = 256;
        const ui32 numBlocks = min((size + blockSize - 1) / blockSize,
                                   TArchProps::MaxBlockCount());
        PoissonRandImpl<<<numBlocks,blockSize, 0, stream>>>(seeds, size, alphas, result);
    }

    __global__ void GaussianRandImpl(ui64* seeds, ui32 seedSize, float* result)
    {
        ui32 i = blockIdx.x * blockDim.x + threadIdx.x;
        while (i < seedSize) {
            ui64 s = seeds[i];
            result[i] = NextNormal(&s);
            seeds[i] = s;
            i += gridDim.x * blockDim.x;
        }
    }

    void GaussianRand(ui64* seeds, ui32 size, float* result, TCudaStream stream)
    {
        const ui32 blockSize = 256;
        const ui32 numBlocks = min((size + blockSize - 1) / blockSize,
                                   TArchProps::MaxBlockCount());
        GaussianRandImpl<<<numBlocks,blockSize, 0, stream>>>(seeds, size, result);
    }

    __global__ void UniformRandImpl(ui64* seeds, ui32 seedSize, float* result)
    {
        ui32 i = blockIdx.x * blockDim.x + threadIdx.x;
        while (i < seedSize) {
            ui64 s = seeds[i];
            result[i] = NextUniform(&s);
            seeds[i] = s;
            i += gridDim.x * blockDim.x;
        }
    }

    void UniformRand(ui64* seeds, ui32 size, float* result, TCudaStream stream)
    {
        const ui32 blockSize = 256;
        const ui32 numBlocks = min((size + blockSize - 1) / blockSize,
                                   TArchProps::MaxBlockCount());
        UniformRandImpl<<<numBlocks, blockSize, 0, stream>>>(seeds, size, result);
    }

    __global__ void GammaRandImpl(ui64* seeds, const float* alphas,
                                  const float* scale, ui32 seedSize, float* result)
    {
        ui32 i = blockIdx.x * blockDim.x + threadIdx.x;
        while (i < seedSize) {
            ui64 s = seeds[i];
            result[i] = NextGamma(&s, alphas[i], scale[i]);
            seeds[i] = s;
            i += gridDim.x * blockDim.x;
        }
    }

    void GammaRand(ui64* seeds, const float* alphas, const float* scale,
                   ui32 size, float* result, TCudaStream stream)
    {
        const ui32 blockSize = 256;
        const ui32 numBlocks = min((size + blockSize - 1) / blockSize,
                                   TArchProps::MaxBlockCount());
        GammaRandImpl<<<numBlocks, blockSize, 0, stream>>>(seeds, alphas, scale, size, result);
    }

    __global__ void BetaRandImpl(ui64* seeds, const float* alphas,
                                 const float* betas, ui32 seedSize, float* result)
    {
        ui32 i = blockIdx.x * blockDim.x + threadIdx.x;
        while (i < seedSize) {
            ui64 s = seeds[i];
            result[i] = NextBeta(&s, alphas[i], betas[i]);
            seeds[i] = s;
            i += gridDim.x * blockDim.x;
        }
    }

    void BetaRand(ui64* seeds, const float* alphas, const float* betas,
                  ui32 size, float* result, TCudaStream stream)
    {
        const ui32 blockSize = 256;
        const ui32 numBlocks = min((size + blockSize - 1) / blockSize,
                                   TArchProps::MaxBlockCount());
        BetaRandImpl<<<numBlocks, blockSize, 0, stream>>>(seeds, alphas, betas, size, result);
    }

}
