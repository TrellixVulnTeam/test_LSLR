#include "hip/hip_runtime.h"
#include "fill.cuh"
#include <catboost/cuda/cuda_lib/kernel/arch.cuh>

namespace NKernel
{

    template<typename T>
    __global__ void FillBufferImpl(T* buffer, T value, ui64  size)
    {
        ui64 i = blockIdx.x * blockDim.x + threadIdx.x;
        while (i < size)
        {
            buffer[i] = value;
            i += gridDim.x * blockDim.x;
        }
    }

    template<typename T>
    void FillBuffer(T* buffer, T value, ui64 size, TCudaStream stream)
    {
        if (size > 0)
        {
            const ui32 blockSize = 512;
            const ui64 numBlocks = min((size + blockSize - 1) / blockSize,
                                         (ui64)TArchProps::MaxBlockCount());
            FillBufferImpl<T> << < numBlocks, blockSize, 0, stream>> > (buffer, value, size);
        }
    }

    template<typename T>
    __global__ void MakeSequenceImpl(T* buffer, ui64  size)
    {
        ui64 i = blockIdx.x * blockDim.x + threadIdx.x;
        while (i < size) {
            buffer[i] = i;
            i += gridDim.x * blockDim.x;
        }
    }

    template<typename T>
    void MakeSequence(T* buffer, ui64  size, TCudaStream stream)
    {
        if (size > 0)
        {
            const ui32 blockSize = 512;
            const ui64 numBlocks = min((size + blockSize - 1) / blockSize,
                                         (ui64)TArchProps::MaxBlockCount());
            MakeSequenceImpl<T> << < numBlocks, blockSize, 0, stream >> > (buffer, size);
        }
    }

    template<typename T>
    __global__ void InversePermutationImpl(const T* indices, T* dst, ui64 size) {
        ui64 i = blockIdx.x * blockDim.x + threadIdx.x;
        while (i < size) {
            dst[indices[i]] = i;
            i += gridDim.x * blockDim.x;
        }
    }

    template<typename T>
    void InversePermutation(const T* order, T* inverseOrder, ui64 size, TCudaStream stream)
    {
        if (size > 0)
        {
            const ui32 blockSize = 512;
            const ui64 numBlocks = min((size + blockSize - 1) / blockSize,
                                       (ui64)TArchProps::MaxBlockCount());
            InversePermutationImpl<T> << < numBlocks, blockSize, 0, stream >> > (order, inverseOrder, size);
        }
    }



    template void FillBuffer<char>(char* buffer, char value, ui64  size, TCudaStream stream);

    template void FillBuffer<unsigned char>(unsigned char* buffer, unsigned char value, ui64  size, TCudaStream stream);

    template void FillBuffer<short>(short* buffer, short value, ui64  size, TCudaStream stream);

    template void FillBuffer<ui16>(ui16* buffer, ui16 value, ui64  size, TCudaStream stream);

    template void FillBuffer<int>(int* buffer, int value, ui64  size, TCudaStream stream);

    template void FillBuffer<ui32>(ui32* buffer, ui32 value, ui64  size, TCudaStream stream);

    template void FillBuffer<float>(float* buffer, float value, ui64  size, TCudaStream stream);

    template void FillBuffer<double>(double* buffer, double value, ui64  size, TCudaStream stream);

    template void FillBuffer<long>(long* buffer, long value, ui64  size, TCudaStream stream);

    template void FillBuffer<ui64>(ui64* buffer, ui64 value, ui64  size, TCudaStream stream);

    template void MakeSequence<int>(int* buffer, ui64  size, TCudaStream stream);

    template void MakeSequence<ui32>(ui32* buffer, ui64  size, TCudaStream stream);

    template void InversePermutation<ui32>(const ui32* order, ui32* inverseOrder, ui64 size, TCudaStream stream);
}
